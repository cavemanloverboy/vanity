#include <stdio.h>
#include <stdint.h>
#include "utils.h"

int num_multi_processors;
int num_blocks;
int num_threads;
int num_gpus;
int max_threads_per_mp;
unsigned long long int target_cycles;

// Get the number of available CUDA devices
extern "C" int get_gpu_count()
{
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess)
    {
        printf("Error getting GPU count: %s\n", hipGetErrorString(error));
        return 0;
    }
    return deviceCount;
}

// Greatest common denominator
// Used in gpu_init() to calculate block_size
int gcd(int a, int b)
{
    return (a == 0) ? b : gcd(b % a, a);
}

// Initializes gpu parameters. Initializes local variables on the host.
// If using multiple (heterogeneous) gpus, this will overwrite device parameters!
extern "C" void gpu_init(int id)
{
    hipDeviceProp_t device_prop;
    int block_size;

    hipError_t cudaerr = hipGetDeviceProperties(&device_prop, id);
    if (cudaerr != hipSuccess)
    {
        printf("getting properties for device failed with error \"%s\".\n", hipGetErrorString(cudaerr));
        exit(EXIT_FAILURE);
    }

    num_threads = device_prop.maxThreadsPerBlock;
    num_multi_processors = device_prop.multiProcessorCount;
    max_threads_per_mp = device_prop.maxThreadsPerMultiProcessor;
    block_size = (max_threads_per_mp / gcd(max_threads_per_mp, num_threads));
    num_threads = 256; // / block_size;
    num_blocks = block_size * num_multi_processors;
    target_cycles = device_prop.clockRate * 1000 * 5; // clockRate is in kHz, mine for 55 seconds
}