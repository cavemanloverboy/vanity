#include "hip/hip_runtime.h"
#include <stdio.h>
#include "base58.h"
#include "vanity.h"
#include "sha256.h"

__device__ int done = 0;
__device__ unsigned long long count = 0;

__device__ bool d_case_insensitive = false;

// TODO:
// 1) Should maybe write a macro for the err handling
// 2) Theoretically possible to reuse device reallocs but it's only one per round so it's kind of ok
extern "C" void vanity_round(
    int id,
    uint8_t *seed,
    uint8_t *base,
    uint8_t *owner,
    char *target,
    uint64_t target_len,
    uint8_t *out,
    bool case_insensitive)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (id >= deviceCount)
    {
        printf("Invalid GPU index: %d\n", id);
        return;
    }

    // Set device and initialize it
    hipSetDevice(id);
    gpu_init(id);

    // Allocate device buffer for seed, base, owner, out, target len, target
    uint8_t *d_buffer;
    hipError_t err = hipMalloc(
        (void **)&d_buffer,
        32               // seed
            + 32         // base
            + 32         // owner
            + 8          // target len
            + target_len // target
            + 16         // out (16 byte seed)
    );
    if (err != hipSuccess)
    {
        printf("CUDA malloc error (d_buffer): %s\n", hipGetErrorString(err));
        return;
    }

    // Copy input seed, base, owner to device
    err = hipMemcpy(d_buffer, seed, 32, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (seed): %s\n", hipGetErrorString(err));
        return;
    }
    err = hipMemcpy(d_buffer + 32, base, 32, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (base): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    err = hipMemcpy(d_buffer + 64, owner, 32, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (owner): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    err = hipMemcpy(d_buffer + 96, &target_len, 8, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (target_len): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    err = hipMemcpy(d_buffer + 104, target, target_len, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (target): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    err = hipMemcpyToSymbol(HIP_SYMBOL(d_case_insensitive), &case_insensitive, 1, 0, hipMemcpyHostToDevice);

    // Reset tracker and counter using hipMemcpyToSymbol
    int zero = 0;
    unsigned long long zero_ull = 0;
    err = hipMemcpyToSymbol(HIP_SYMBOL(done), &zero, sizeof(int));
    if (err != hipSuccess)
    {
        printf("CUDA memcpy to symbol error (done): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    err = hipMemcpyToSymbol(HIP_SYMBOL(count), &zero_ull, sizeof(unsigned long long));
    if (err != hipSuccess)
    {
        printf("CUDA memcpy to symbol error (count): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    // Launch vanity search kernel
    vanity_search<<<num_blocks, num_threads>>>(d_buffer, num_blocks * num_threads);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA launch error: %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    // Copy result to host
    err = hipMemcpy(out, d_buffer + 104 + target_len, 16, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (d_out): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    err = hipMemcpyFromSymbol(out + 16, HIP_SYMBOL(count), 8, 0, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (count): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    // Free pointers
    hipFree(d_buffer);
}

__device__ uint8_t const alphanumeric[63] = "0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz";

__global__ void
vanity_search(uint8_t *buffer, uint64_t stride)
{
    // Deconstruct buffer
    uint8_t *seed = buffer;
    uint8_t *base = buffer + 32;
    uint8_t *owner = buffer + 64;
    uint64_t target_len;
    memcpy(&target_len, buffer + 96, 8);
    uint8_t *target = buffer + 104;
    uint8_t *out = (buffer + 104 + target_len);

    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned char local_out[32] = {0};
    unsigned char local_encoded[44] = {0};
    uint64_t local_seed[4];

    // Pseudo random generator
    CUDA_SHA256_CTX ctx;
    cuda_sha256_init(&ctx);
    cuda_sha256_update(&ctx, (BYTE *)(seed), 32);
    cuda_sha256_update(&ctx, (BYTE *)(&idx), 8);
    cuda_sha256_final(&ctx, (BYTE *)local_seed);

    CUDA_SHA256_CTX address_sha;
    cuda_sha256_init(&address_sha);
    cuda_sha256_update(&address_sha, (BYTE *)base, 32);

    for (uint64_t iter = 0; iter < 1000 * 1000 * 1000 * 1000; iter++)
    {
        // Has someone found a result?
        if (iter % 100 == 0)
        {
            if (atomicMax(&done, 0) == 1)
            {
                atomicAdd(&count, iter);
                return;
            }
        }

        cuda_sha256_init(&ctx);
        cuda_sha256_update(&ctx, (BYTE *)local_seed, 16);
        cuda_sha256_final(&ctx, (BYTE *)local_seed);

        uint32_t *indices = (uint32_t *)&local_seed;
        uint8_t create_account_seed[16] = {
            alphanumeric[indices[0] % 62],
            alphanumeric[indices[1] % 62],
            alphanumeric[indices[2] % 62],
            alphanumeric[indices[3] % 62],
            alphanumeric[indices[4] % 62],
            alphanumeric[indices[5] % 62],
            alphanumeric[indices[6] % 62],
            alphanumeric[indices[7] % 62],
            alphanumeric[(indices[0] >> 2) % 62],
            alphanumeric[(indices[1] >> 2) % 62],
            alphanumeric[(indices[2] >> 2) % 62],
            alphanumeric[(indices[3] >> 2) % 62],
            alphanumeric[(indices[4] >> 2) % 62],
            alphanumeric[(indices[5] >> 2) % 62],
            alphanumeric[(indices[6] >> 2) % 62],
            alphanumeric[(indices[7] >> 2) % 62],
        };

        // Calculate and encode public
        CUDA_SHA256_CTX address_sha_local;
        memcpy(&address_sha_local, &address_sha, sizeof(CUDA_SHA256_CTX));
        cuda_sha256_update(&address_sha_local, (BYTE *)create_account_seed, 16);
        cuda_sha256_update(&address_sha_local, (BYTE *)owner, 32);
        cuda_sha256_final(&address_sha_local, (BYTE *)local_out);
        fd_base58_encode_32(local_out, (unsigned char *)(&local_encoded), d_case_insensitive);

        // Check target
        if (matches_target((unsigned char *)local_encoded, (unsigned char *)target, target_len))
        {
            // Are we first to write result?
            if (atomicMax(&done, 1) == 0)
            {
                // seed for CreateAccountWithSeed
                memcpy(out, create_account_seed, 16);
            }

            atomicAdd(&count, iter + 1);
            return;
        }
    }
}

__device__ bool matches_target(unsigned char *a, unsigned char *target, uint64_t n)
{
    for (int i = 0; i < n; i++)
    {
        if (a[i] != target[i])
            return false;
    }
    return true;
}