#include "hip/hip_runtime.h"
#include <stdio.h>
#include "base58.h"
#include "vanity.h"
#include "sha256.h"

__device__ int done = 0;
__device__ unsigned long long count = 0;
__device__ bool d_case_insensitive = false;
__device__ bool d_leet_speak = false;

// TODO:
// 1) Should maybe write a macro for the err handling
// 2) Theoretically possible to reuse device reallocs but it's only one per round so it's kind of ok
extern "C" void vanity_round(
    int id,
    uint8_t *seed,
    uint8_t *base,
    uint8_t *owner,
    char *prefix,
    uint64_t prefix_len,
    char *suffix,
    uint64_t suffix_len,
    char *any,
    uint64_t any_len,
    uint8_t *out,
    bool case_insensitive,
    bool leet_speak)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (id >= deviceCount)
    {
        printf("Invalid GPU index: %d\n", id);
        return;
    }

    // Set device and initialize it
    hipSetDevice(id);
    gpu_init(id);

    // Calculate new buffer size including 'any' string
    uint8_t *d_buffer;
    hipError_t err = hipMalloc(
        (void **)&d_buffer,
        32               // seed
            + 32         // base
            + 32         // owner
            + 8          // prefix len
            + prefix_len // prefix
            + 8          // suffix len
            + suffix_len // suffix
            + 8          // any len
            + any_len    // any string
            + 16         // out (16 byte seed)
    );
    printf("CUDA device count: %d\n", deviceCount);
    printf("Setting GPU device %d\n", id);
    printf("CUDA malloc successful for d_buffer\n");

    if (err != hipSuccess)
    {
        printf("CUDA malloc error (d_buffer): %s\n", hipGetErrorString(err));
        return;
    }

    // Copy input seed, base, owner to device
    err = hipMemcpy(d_buffer, seed, 32, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (seed): %s\n", hipGetErrorString(err));
        return;
    }
    err = hipMemcpy(d_buffer + 32, base, 32, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (base): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    err = hipMemcpy(d_buffer + 64, owner, 32, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (owner): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    err = hipMemcpy(d_buffer + 96, &prefix_len, 8, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (prefix_len): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    // Copy prefix to device memory
    err = hipMemcpy(d_buffer + 104, prefix, prefix_len, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (prefix): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    // Copy suffix length to device memory
    err = hipMemcpy(d_buffer + 104 + prefix_len, &suffix_len, 8, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (suffix_len): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    // Copy suffix to device memory
    err = hipMemcpy(d_buffer + 112 + prefix_len, suffix, suffix_len, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (suffix): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    // Copy any length and string
    err = hipMemcpy(d_buffer + 112 + prefix_len + suffix_len, &any_len, 8, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (any_len): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    if (any_len > 0)
    {
        err = hipMemcpy(d_buffer + 120 + prefix_len + suffix_len, any, any_len, hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            printf("CUDA memcpy error (any): %s\n", hipGetErrorString(err));
            hipFree(d_buffer);
            return;
        }
    }

    err = hipMemcpyToSymbol(HIP_SYMBOL(d_case_insensitive), &case_insensitive, 1, 0, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy to symbol error (done): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    // Reset tracker and counter using hipMemcpyToSymbol
    int zero = 0;
    unsigned long long zero_ull = 0;
    err = hipMemcpyToSymbol(HIP_SYMBOL(done), &zero, sizeof(int));
    if (err != hipSuccess)
    {
        printf("CUDA memcpy to symbol error (done): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    err = hipMemcpyToSymbol(HIP_SYMBOL(count), &zero_ull, sizeof(unsigned long long));
    if (err != hipSuccess)
    {
        printf("CUDA memcpy to symbol error (count): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    // Copy leet_speak setting to device
    err = hipMemcpyToSymbol(HIP_SYMBOL(d_leet_speak), &leet_speak, sizeof(bool));
    if (err != hipSuccess)
    {
        printf("CUDA memcpy to symbol error (leet_speak): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    printf("Launching vanity_search kernel\n");
    // Launch vanity search kernel
    vanity_search<<<num_blocks, num_threads>>>(d_buffer, num_blocks * num_threads);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA launch error: %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    printf("Vanity search kernel launched and synchronized\n");

    // Copy result to host
    // Copy the output from device to host
    err = hipMemcpy(out, d_buffer + 104 + prefix_len + suffix_len, 16, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (d_out): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    // Copy the 'count' value from the device to host
    err = hipMemcpyFromSymbol(out + 16, HIP_SYMBOL(count), 8, 0, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (count): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    // Free pointers
    hipFree(d_buffer);
}

__device__ uint8_t const alphanumeric[63] = "0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz";

__global__ void
vanity_search(uint8_t *buffer, uint64_t stride)
{
    // Deconstruct buffer
    uint8_t *seed = buffer;
    uint8_t *base = buffer + 32;
    uint8_t *owner = buffer + 64;
    uint64_t prefix_len;
    uint64_t suffix_len;
    uint64_t any_len;

    // Get the lengths from buffer
    memcpy(&prefix_len, buffer + 96, 8);
    memcpy(&suffix_len, buffer + 104 + prefix_len, 8);
    memcpy(&any_len, buffer + 112 + prefix_len + suffix_len, 8);

    // Get the pointers to the strings
    uint8_t *prefix = buffer + 104;
    uint8_t *suffix = buffer + 112 + prefix_len;
    uint8_t *any = buffer + 120 + prefix_len + suffix_len;
    uint8_t *out = buffer + 120 + prefix_len + suffix_len + any_len;

    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned char local_out[32] = {0};
    unsigned char local_encoded[44] = {0};
    uint64_t local_seed[4];

    // Pseudo random generator
    CUDA_SHA256_CTX ctx;
    cuda_sha256_init(&ctx);
    cuda_sha256_update(&ctx, (BYTE *)(seed), 32);
    cuda_sha256_update(&ctx, (BYTE *)(&idx), 8);
    cuda_sha256_final(&ctx, (BYTE *)local_seed);

    CUDA_SHA256_CTX address_sha;
    cuda_sha256_init(&address_sha);
    cuda_sha256_update(&address_sha, (BYTE *)base, 32);

    for (uint64_t iter = 0; iter < 1000 * 1000 * 1000; iter++)
    {
        // Has someone found a result?
        if (iter % 100 == 0)
        {
            if (atomicMax(&done, 0) == 1)
            {
                atomicAdd(&count, iter);
                return;
            }
        }

        cuda_sha256_init(&ctx);
        cuda_sha256_update(&ctx, (BYTE *)local_seed, 16);
        cuda_sha256_final(&ctx, (BYTE *)local_seed);

        uint32_t *indices = (uint32_t *)&local_seed;
        uint8_t create_account_seed[16] = {
            alphanumeric[indices[0] % 62],
            alphanumeric[indices[1] % 62],
            alphanumeric[indices[2] % 62],
            alphanumeric[indices[3] % 62],
            alphanumeric[indices[4] % 62],
            alphanumeric[indices[5] % 62],
            alphanumeric[indices[6] % 62],
            alphanumeric[indices[7] % 62],
            alphanumeric[(indices[0] >> 2) % 62],
            alphanumeric[(indices[1] >> 2) % 62],
            alphanumeric[(indices[2] >> 2) % 62],
            alphanumeric[(indices[3] >> 2) % 62],
            alphanumeric[(indices[4] >> 2) % 62],
            alphanumeric[(indices[5] >> 2) % 62],
            alphanumeric[(indices[6] >> 2) % 62],
            alphanumeric[(indices[7] >> 2) % 62],
        };

        // Calculate and encode public
        CUDA_SHA256_CTX address_sha_local;
        memcpy(&address_sha_local, &address_sha, sizeof(CUDA_SHA256_CTX));
        cuda_sha256_update(&address_sha_local, (BYTE *)create_account_seed, 16);
        cuda_sha256_update(&address_sha_local, (BYTE *)owner, 32);
        cuda_sha256_final(&address_sha_local, (BYTE *)local_out);
        fd_base58_encode_32(local_out, (unsigned char *)(&local_encoded), d_case_insensitive);

        // Check prefix and suffix
        // printf("Got key: %s\n", local_encoded);

        if (matches_search(
                (unsigned char *)local_encoded,
                (unsigned char *)prefix,
                prefix_len,
                (unsigned char *)suffix,
                suffix_len,
                (unsigned char *)any,
                any_len))
        {
            // Are we first to write result?
            if (atomicMax(&done, 1) == 0)
            {
                // seed for CreateAccountWithSeed
                // printf("Match found! Copying result to out\n");

                memcpy(out, create_account_seed, 16);
            }

            atomicAdd(&count, iter + 1);
            return;
        }
    }
}

__device__ bool chars_match_leet(char a, char b)
{
    if (a == b)
        return true;

    switch (a)
    {
    case 'a':
    case 'A':
        return b == '4';
    case 'e':
    case 'E':
        return b == '3';
    case 't':
    case 'T':
        return b == '7';
    case 'l':
    case 'L':
        return b == '1';
    case 'i':
    case 'I':
        return b == '1';
    case 's':
    case 'S':
        return b == '5';
    case 'g':
    case 'G':
        return b == '6';
    case 'b':
    case 'B':
        return b == '8';
    case 'z':
    case 'Z':
        return b == '2';
    }

    switch (b)
    {
    case '4':
        return a == 'a' || a == 'A';
    case '3':
        return a == 'e' || a == 'E';
    case '7':
        return a == 't' || a == 'T';
    case '1':
        return a == 'l' || a == 'L' || a == 'i' || a == 'I';
    case '5':
        return a == 's' || a == 'S';
    case '6':
        return a == 'g' || a == 'G';
    case '8':
        return a == 'b' || a == 'B';
    case '2':
        return a == 'z' || a == 'Z';
    }

    return false;
}

__device__ bool matches_search(
    unsigned char *a,
    unsigned char *prefix,
    uint64_t prefix_len,
    unsigned char *suffix,
    uint64_t suffix_len,
    unsigned char *any,
    uint64_t any_len)
{
    bool prefix_matches = true;
    bool suffix_matches = true;
    bool any_matches = true;

    // Print full address and search criteria
    printf("\nCUDA Checking address: %s\n", a);
    if (prefix_len > 0)
        printf("Looking for prefix: %.*s (len=%lu)\n", (int)prefix_len, prefix, prefix_len);
    if (suffix_len > 0)
        printf("Looking for suffix: %.*s (len=%lu)\n", (int)suffix_len, suffix, suffix_len);
    if (any_len > 0)
        printf("Looking for any: %.*s (len=%lu)\n", (int)any_len, any, any_len);
    printf("Leet speak: %s\n", d_leet_speak ? "enabled" : "disabled");

    // Skip checks if length is 0
    if (prefix_len > 0)
    {
        // Check prefix
        for (int i = 0; i < prefix_len; i++)
        {
            if (d_leet_speak)
            {
                if (!chars_match_leet(prefix[i], a[i]))
                    prefix_matches = false;
            }
            else if (a[i] != prefix[i])
            {
                prefix_matches = false;
            }
        }
    }

    if (suffix_len > 0)
    {
        // Check suffix
        for (int i = 0; i < suffix_len; i++)
        {
            if (d_leet_speak)
            {
                if (!chars_match_leet(suffix[i], a[44 - suffix_len + i]))
                    suffix_matches = false;
            }
            else if (a[44 - suffix_len + i] != suffix[i])
            {
                suffix_matches = false;
            }
        }
    }

    if (any_len > 0)
    {
        // Check for 'any' string anywhere in the address
        bool found = false;
        for (int i = 0; i <= 44 - any_len; i++)
        {
            bool match = true;
            for (int j = 0; j < any_len; j++)
            {
                if (d_leet_speak)
                {
                    if (!chars_match_leet(any[j], a[i + j]))
                    {
                        match = false;
                        break;
                    }
                }
                else if (a[i + j] != any[j])
                {
                    match = false;
                    break;
                }
            }
            if (match)
            {
                found = true;
                break;
            }
        }
        if (!found)
            any_matches = false;
    }

    if (prefix_matches && suffix_matches && any_matches)
    {
        printf("\nCUDA MATCH FOUND!\n");
        printf("Full address: %s\n", a);
        printf("Prefix match: %s\n", prefix_matches ? "YES" : "NO");
        printf("Suffix match: %s\n", suffix_matches ? "YES" : "NO");
        printf("Any match: %s\n", any_matches ? "YES" : "NO");
    }

    return prefix_matches && suffix_matches && any_matches;
}
