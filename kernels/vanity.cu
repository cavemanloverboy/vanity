#include "hip/hip_runtime.h"
#include <stdio.h>
#include "base58.h"
#include "vanity.h"
#include "sha256.h"

__device__ int done = 0;
__device__ unsigned long long count = 0;
__device__ bool d_case_insensitive = false;

// TODO:
// 1) Should maybe write a macro for the err handling
// 2) Theoretically possible to reuse device reallocs but it's only one per round so it's kind of ok
extern "C" void vanity_round(
    int id,
    uint8_t *seed,
    uint8_t *base,
    uint8_t *owner,
    char *prefix,
    uint64_t prefix_len,
    char *suffix,
    uint64_t suffix_len,
    uint8_t *out,
    bool case_insensitive)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (id >= deviceCount)
    {
        printf("Invalid GPU index: %d\n", id);
        return;
    }


    // Set device and initialize it
    hipSetDevice(id);
    gpu_init(id);


    // Allocate device buffer for seed, base, owner, out, prefix len, prefix, suffix_len, suffix
    uint8_t *d_buffer;
    hipError_t err = hipMalloc(
        (void **)&d_buffer,
        32               // seed
            + 32         // base
            + 32         // owner
            + 8          // prefix len
            + prefix_len // prefix
            + 8          // suffix len
            + suffix_len // suffix
            + 16         // out (16 byte seed)
    );
    printf("CUDA device count: %d\n", deviceCount);
    printf("Setting GPU device %d\n", id);
    printf("CUDA malloc successful for d_buffer\n");

    if (err != hipSuccess)
    {
        printf("CUDA malloc error (d_buffer): %s\n", hipGetErrorString(err));
        return;
    }

    // Copy input seed, base, owner to device
    err = hipMemcpy(d_buffer, seed, 32, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (seed): %s\n", hipGetErrorString(err));
        return;
    }
    err = hipMemcpy(d_buffer + 32, base, 32, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (base): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    err = hipMemcpy(d_buffer + 64, owner, 32, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (owner): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    err = hipMemcpy(d_buffer + 96, &prefix_len, 8, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (prefix_len): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    
    // Copy prefix to device memory
    err = hipMemcpy(d_buffer + 104, prefix, prefix_len, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (prefix): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    
    // Copy suffix length to device memory
    err = hipMemcpy(d_buffer + 104 + prefix_len, &suffix_len, 8, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (suffix_len): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    
    // Copy suffix to device memory
    err = hipMemcpy(d_buffer + 112 + prefix_len, suffix, suffix_len, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (suffix): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    err = hipMemcpyToSymbol(HIP_SYMBOL(d_case_insensitive), &case_insensitive, 1, 0, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy to symbol error (done): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    
    // Reset tracker and counter using hipMemcpyToSymbol
    int zero = 0;
    unsigned long long zero_ull = 0;
    err = hipMemcpyToSymbol(HIP_SYMBOL(done), &zero, sizeof(int));
    if (err != hipSuccess)
    {
        printf("CUDA memcpy to symbol error (done): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }
    err = hipMemcpyToSymbol(HIP_SYMBOL(count), &zero_ull, sizeof(unsigned long long));
    if (err != hipSuccess)
    {
        printf("CUDA memcpy to symbol error (count): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    printf("Launching vanity_search kernel\n");
    // Launch vanity search kernel
    vanity_search<<<num_blocks, num_threads>>>(d_buffer, num_blocks * num_threads);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA launch error: %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    printf("Vanity search kernel launched and synchronized\n");

    // Copy result to host
    // Copy the output from device to host
    err = hipMemcpy(out, d_buffer + 104 + prefix_len + suffix_len, 16, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (d_out): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }

    // Copy the 'count' value from the device to host
    err = hipMemcpyFromSymbol(out + 16, HIP_SYMBOL(count), 8, 0, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("CUDA memcpy error (count): %s\n", hipGetErrorString(err));
        hipFree(d_buffer);
        return;
    }



    // Free pointers
    hipFree(d_buffer);

}

__device__ uint8_t const alphanumeric[63] = "0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz";

__global__ void
vanity_search(uint8_t *buffer, uint64_t stride)
{
    // Deconstruct buffer
    uint8_t *seed = buffer;
    uint8_t *base = buffer + 32;
    uint8_t *owner = buffer + 64;
    uint64_t prefix_len;
    uint64_t suffix_len;
    
    // Assuming the prefix and suffix lengths are already in the buffer
    memcpy(&prefix_len, buffer + 96, 8);
    memcpy(&suffix_len, buffer + 104 + prefix_len, 8);  // Assuming suffix_len is placed after the prefix
    
    uint8_t *prefix = buffer + 104;  // The prefix starts after the prefix_len
    uint8_t *suffix = buffer + 104 + prefix_len;  // The suffix starts after the prefix data
    uint8_t *out = (buffer + 104 + prefix_len + suffix_len);  // Out is after both prefix and suffix
    

    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned char local_out[32] = {0};
    unsigned char local_encoded[44] = {0};
    uint64_t local_seed[4];


    // Pseudo random generator
    CUDA_SHA256_CTX ctx;
    cuda_sha256_init(&ctx);
    cuda_sha256_update(&ctx, (BYTE *)(seed), 32);
    cuda_sha256_update(&ctx, (BYTE *)(&idx), 8);
    cuda_sha256_final(&ctx, (BYTE *)local_seed);

    CUDA_SHA256_CTX address_sha;
    cuda_sha256_init(&address_sha);
    cuda_sha256_update(&address_sha, (BYTE *)base, 32);

    for (uint64_t iter = 0; iter < 1000 * 1000 * 1000; iter++)
    {
        // Has someone found a result?
        if (iter % 100 == 0)
        {
            if (atomicMax(&done, 0) == 1)
            {
                atomicAdd(&count, iter);
                return;
            }
        }

        cuda_sha256_init(&ctx);
        cuda_sha256_update(&ctx, (BYTE *)local_seed, 16);
        cuda_sha256_final(&ctx, (BYTE *)local_seed);

        uint32_t *indices = (uint32_t *)&local_seed;
        uint8_t create_account_seed[16] = {
            alphanumeric[indices[0] % 62],
            alphanumeric[indices[1] % 62],
            alphanumeric[indices[2] % 62],
            alphanumeric[indices[3] % 62],
            alphanumeric[indices[4] % 62],
            alphanumeric[indices[5] % 62],
            alphanumeric[indices[6] % 62],
            alphanumeric[indices[7] % 62],
            alphanumeric[(indices[0] >> 2) % 62],
            alphanumeric[(indices[1] >> 2) % 62],
            alphanumeric[(indices[2] >> 2) % 62],
            alphanumeric[(indices[3] >> 2) % 62],
            alphanumeric[(indices[4] >> 2) % 62],
            alphanumeric[(indices[5] >> 2) % 62],
            alphanumeric[(indices[6] >> 2) % 62],
            alphanumeric[(indices[7] >> 2) % 62],
        };


        // Calculate and encode public
        CUDA_SHA256_CTX address_sha_local;
        memcpy(&address_sha_local, &address_sha, sizeof(CUDA_SHA256_CTX));
        cuda_sha256_update(&address_sha_local, (BYTE *)create_account_seed, 16);
        cuda_sha256_update(&address_sha_local, (BYTE *)owner, 32);
        cuda_sha256_final(&address_sha_local, (BYTE *)local_out);
        fd_base58_encode_32(local_out, (unsigned char *)(&local_encoded), d_case_insensitive);

        // Check prefix and suffix
        // printf("Got key: %s\n", local_encoded);

        if (matches_search((unsigned char *)local_encoded, (unsigned char *)prefix, prefix_len, (unsigned char *)suffix, suffix_len))
        {
            // Are we first to write result?
            if (atomicMax(&done, 1) == 0)
            {
                // seed for CreateAccountWithSeed
                // printf("Match found! Copying result to out\n");

                memcpy(out, create_account_seed, 16);
            }

            atomicAdd(&count, iter + 1);
            return;
        }
    }
}

__device__ bool matches_search(unsigned char *a, unsigned char *prefix, uint64_t prefix_len, unsigned char *suffix, uint64_t suffix_len)
{
    for (int i = 0; i < prefix_len; i++) 
    {
        if (a[i] != prefix[i])
            return false;
    }

    for (int i = 0; i < suffix_len; i++) 
    {
        if (a[prefix_len + i] != suffix[i])
            return false;
    }

    return true;
}
